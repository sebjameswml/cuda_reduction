#include "hip/hip_runtime.h"
#include <cmath>
#include <limits>
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h> // for float3 type (a struct of 3 floats)

// Parameters

// How many thread blocks will we sum up?
static constexpr int arrayblocks = 10240;
// How many threads per block to specify.
static constexpr int threadsperblock = 512;

// In each warp reduce three values per thread
__inline__ __device__ float3 warpReduceSum (float valR, float valG, float valB)
{
    for (int offset = warpSize/2; offset > 0; offset >>= 1) {
        valR += __shfl_down_sync(0xffffffff, valR, offset);
        valG += __shfl_down_sync(0xffffffff, valG, offset);
        valB += __shfl_down_sync(0xffffffff, valB, offset);
    }
    return make_float3 (valR, valG, valB);
}

__inline__ __device__ float3 blockReduceSum (float3 val)
{
    static __shared__ float3 shared[32];    // Shared mem for 32 partial sums
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;
    val = warpReduceSum (val.x, val.y, val.z); // Each warp performs partial reduction
    if (lane == 0) { shared[wid] = val; }  // Write reduced value to shared memory
    __syncthreads();                       // Wait for all partial reductions
    // read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : make_float3(0.0f, 0.0f, 0.0f);
    if (wid == 0) { val = warpReduceSum (val.x, val.y, val.z); } // Final reduce within first warp
    return val;
}

// Input is float3 format.
__global__ void reduceit (float3* in, float3* out, int N)
{
    float3 sum = make_float3(0.0f, 0.0f, 0.0f);
    // reduce multiple elements per thread
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        sum.x += in[i].x;
        sum.y += in[i].y;
        sum.z += in[i].z;
    }
    sum = blockReduceSum (sum);
    if (threadIdx.x == 0) { out[blockIdx.x] = sum; }
}

__host__ void shufflesum_gpu_work (float3* in, float3* out, int N)
{
    int threads = threadsperblock;
    int blocks = min((N + threads - 1) / threads, 1024);
    reduceit<<<blocks, threads>>>(in, out, N);
    reduceit<<<1, 1024>>>(out, out, blocks);
}

__host__ float3 shufflesum_gpu (float3* d_weight_ar, int arraysz)
{
    float3* d_scanf_ar = nullptr;
    hipMalloc (&d_scanf_ar, arraysz * 3 * sizeof(float));
    shufflesum_gpu_work (d_weight_ar, d_scanf_ar, arraysz);
    float3 sum;
    hipMemcpy(&sum, d_scanf_ar, 3 * sizeof(float), hipMemcpyDeviceToHost);
    return sum;
}

int main()
{
    int arraysz = arrayblocks * threadsperblock;

    std::vector<float3> weight_ar (arraysz, make_float3(0.0f, 0.0f, 0.0f));

    // Now some non-zero, non-unary weights
    for (int i = 0; i < arraysz; ++i) {
        if (i % 2 == 0) {
            weight_ar[i] = make_float3 (2.6f, 1.03f, 3.4f);
        } else if ((i % 3) == 0) {
            weight_ar[i] = make_float3 (2.73f, 2.03f, 2.4f);
        } else if ((i % 4) == 0) {
            weight_ar[i] = make_float3 (-3.73f, 3.03f, 1.4f);
        } else if ((i % 5) == 0) {
            weight_ar[i] = make_float3 (1.73f, 4.03f, 0.4f);
        } else if ((i % 6) == 0) {
            weight_ar[i] = make_float3 (-3.02f, 1.03f, -1.4f);
        }
    }

    float3 cpu_sum = make_float3 (0.0f, 0.0f, 0.0f);
    for (auto w : weight_ar) {
        cpu_sum.x += w.x;
        cpu_sum.y += w.y;
        cpu_sum.z += w.z;
    }

    // Copy to GPU memory:
    float3* d_weight_ar = nullptr;
    hipMalloc (&d_weight_ar, arraysz * 3 * sizeof(float));
    hipMemcpy (d_weight_ar, weight_ar.data(), arraysz * 3 * sizeof(float), hipMemcpyHostToDevice);

    // Call the function:
    float3 gpu_sum = shufflesum_gpu (d_weight_ar, arraysz);

    std::cout << "GPU array sum is (" << gpu_sum.x << "," << gpu_sum.y << "," << gpu_sum.z << ")."
              << " CPU array sum is " << cpu_sum.x << "," << cpu_sum.y << "," << cpu_sum.z << "\n";

    return 0;
}
