#include "hip/hip_runtime.h"
#include <cmath>
#include <limits>
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>

// Parameters

// How many thread blocks will we sum up?
static constexpr int arrayblocks = 2;
// How many threads per block to specify.
static constexpr int threadsperblock = 512;
// Mask for __shfl_down_sync
static constexpr unsigned int all_in_warp = 0xffffffff;

// T __shfl_down_sync(unsigned mask, T var, unsigned int delta, int width=warpSize);
__inline__ __device__ float warpReduceSum (float val)
{
    for (int offset = warpSize/2; offset > 0; offset >>= 1) {
        val += __shfl_down_sync (all_in_warp, val, offset);
    }
    return val;
}

__inline__ __device__ float blockReduceSum (float val)
{
    static __shared__ float shared[32];    // Shared mem for 32 partial sums
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;
    val = warpReduceSum (val);             // Each warp performs partial reduction
    if (lane == 0) { shared[wid] = val; }  // Write reduced value to shared memory
    __syncthreads();                       // Wait for all partial reductions
    // read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0.0f;
    if (wid == 0) { val = warpReduceSum (val); } // Final reduce within first warp
    return val;
}

__global__ void reduceit (float *in, float* out, int N)
{
    float sum = 0.0f;
    // reduce multiple elements per thread
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        sum += in[i];
    }
    sum = blockReduceSum (sum);
    if (threadIdx.x == 0) { out[blockIdx.x] = sum; }
}

__host__ void shufflesum_gpu_work (float *d_in, float* d_out, int N)
{
    int blocks = min((N + threadsperblock - 1) / threadsperblock, 1024);
    reduceit<<<blocks, threadsperblock>>>(d_in, d_out, N);
    // Debug. Copy data in out and examine.
    std::vector<float> r_in (N, 0.0f);
    hipMemcpy (r_in.data(), d_in, N * sizeof(float), hipMemcpyDeviceToHost);
    std::vector<float> r_out (N, 0.0f);
    hipMemcpy (r_out.data(), d_out, N * sizeof(float), hipMemcpyDeviceToHost);
#define DEBUG_FIRST_REDUCTION_KERNEL 1 // reduce arrayblocks to 2!
#ifdef DEBUG_FIRST_REDUCTION_KERNEL
    // After the above, out contains arrayblocks values (everything else is 0)
    for (int i = 0; i < N; ++i) {
        std::cout << "After first reduceit in["<<i<<"] = " << r_in[i] << ", out = " << r_out[i] << std::endl;
    }
#endif
    reduceit<<<1, 1024>>>(d_out, d_out, blocks);
    // After this reduction, only out[0] contains anything
}

__host__ float shufflesum_gpu (float* d_in, int arraysz)
{
    float* d_out = nullptr;
    hipMalloc (&d_out, arraysz * sizeof(float));
    shufflesum_gpu_work (d_in, d_out, arraysz);
    float sum;
    hipMemcpy(&sum, d_out, sizeof(float), hipMemcpyDeviceToHost);

    // Can free

    return sum;
}

int main()
{
    int arraysz = arrayblocks * threadsperblock;
    std::cout << "Array size is " << arraysz << std::endl;
    std::cout << "arraysz * epsilon is " << arraysz * std::numeric_limits<float>::epsilon() << std::endl;

    std::vector<float> weight_ar (arraysz, 0.0f);

    // Now some non-zero, non-unary weights
    for (int i = 0; i < arraysz/2; ++i) {
        if (i % 2 == 0) {
            weight_ar[i] = 0.0032f;
        } else if ((i % 3) == 0) {
            weight_ar[i] = 0.00021f;
        } else if ((i % 4) == 0) {
            weight_ar[i] = -0.032f;
        } else if ((i % 5) == 0) {
            weight_ar[i] = 0.0051f;
        } else if ((i % 6) == 0) {
            weight_ar[i] = -0.000435f;
        }
        // Top half same as bottom half
        weight_ar[i + arraysz/2] = weight_ar[i];
    }


    float cpu_sum = 0.0f;
    for (auto w : weight_ar) { cpu_sum += w; }

    // Copy to GPU memory:
    float* d_weight_ar = nullptr;
    hipMalloc (&d_weight_ar, arraysz * sizeof(float));
    hipMemcpy (d_weight_ar, weight_ar.data(), arraysz * sizeof(float), hipMemcpyHostToDevice);

    // Call the function:
    float gpu_sum = shufflesum_gpu (d_weight_ar, arraysz);

    std::cout << "GPU array sum is " << gpu_sum << ". CPU array sum is " << cpu_sum
              << "\nGPU/CPU method difference is "
              << (std::abs(gpu_sum - cpu_sum) / std::numeric_limits<float>::epsilon())
              << " epsilons = " << std::abs(gpu_sum - cpu_sum) << "\n";

    return 0;
}
