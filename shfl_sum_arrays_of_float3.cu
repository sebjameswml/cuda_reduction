#include <cmath>
#include <limits>
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h> // for float3 type (a struct of 3 floats)

// Parameters

// How many arrays to sum?
static constexpr int number_of_arrays = 32;
// How many elements per array? May be > or < threadsperblock
static constexpr int elements_per_array = 1024;
// Ideal number of threads per block
static constexpr int threadsperblock = 512;
// Mask for __shfl_down_sync
static constexpr unsigned int all_in_warp = 0xffffffff;

// In each warp reduce three values per thread
__inline__ __device__ float3 warpReduceSum (float valR, float valG, float valB)
{
    for (int offset = warpSize/2; offset > 0; offset >>= 1) {
        valR += __shfl_down_sync(all_in_warp, valR, offset);
        valG += __shfl_down_sync(all_in_warp, valG, offset);
        valB += __shfl_down_sync(all_in_warp, valB, offset);
    }
    return make_float3 (valR, valG, valB);
}

// Run by the 32 threads of a warp
__inline__ __device__ float3 blockReduceSum (float3 val)
{
    static __shared__ float3 shared[32];    // Shared mem for 32 partial sums
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;
    val = warpReduceSum (val.x, val.y, val.z); // Each warp performs partial reduction
    if (lane == 0) { shared[wid] = val; }  // Write reduced value to shared memory
    __syncthreads();                       // Wait for all partial reductions
    // read from shared memory only if that warp existed

    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : make_float3(0.0f, 0.0f, 0.0f);

    if (wid == 0) { val = warpReduceSum (val.x, val.y, val.z); } // Final reduce within first warp
    return val;
}

// Input is float3 format.
__global__ void reduceit_arrays (float3* in, float3* out, int n_arrays, int n_elements)
{
    float3 sum = make_float3(0.0f, 0.0f, 0.0f);
    // The y axis of our threads/threadblocks indexes which of the n_arrays this sum relates to
    int omm_id = blockIdx.y * blockDim.y + threadIdx.y;
    // This gives a memory offset to get to the right part of the input memory
    int mem_offset = omm_id * n_elements;
    // Number of sums is the number of 1D threadblocks that span n_elements. This is gridDim.x.
    int n_sums = gridDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n_elements && omm_id < n_arrays;
         i += blockDim.x * gridDim.x) {
        sum.x += in[mem_offset + i].x;
        sum.y += in[mem_offset + i].y;
        sum.z += in[mem_offset + i].z;
    }

    sum = blockReduceSum (sum);
    __syncthreads();

    // This gets the correct output location in out.
    if (threadIdx.x == 0 && omm_id < n_arrays) {
        out[omm_id * n_sums + blockIdx.x] = sum;
    }
}

// Compute the sum of each of N-arrays arrays that are layed out one-after-another in the input
// in. out should be the same size as in and is used when shuffle-computing the sums and also to
// hold the results.
__host__ void shufflesum_arrays (float3* in, int n_arrays, int n_elements,
                                 std::vector<float3>& sums, std::vector<float3>& final_sums)
{
    // Working out the threads per block is the thing

    // threadsperblock is the ideal size (512). warp size is 32.  So basic threadblock
    // size should be 32 in x and 16 in y, giving 512 threads. If n_elements < 32 or
    // n_arrays < 16, then some kind of padding should happen so that this works, even
    // if it's slow.  EXCEPT for the reduction, I can't mix memory values from different
    // arrays in a threadblock, so the threadblock has to be 1D and thus configurable -
    // dynamically sized to match the number of elements in the array.
    int warps_base = n_elements / 32;
    int warps_extra = n_elements % 32;
    int tbx = (warps_base * 32) + (warps_extra ? 32 : 0);
    tbx = std::min (tbx, threadsperblock);
    dim3 stg1_blockdim(tbx, 1);

    // Then figure out how many threadblocks to run.
    dim3 stg1_griddim(1, 1);
    stg1_griddim.x = n_elements / stg1_blockdim.x + (n_elements % stg1_blockdim.x ? 1 : 0);
    stg1_griddim.y = n_arrays / stg1_blockdim.y + (n_arrays % stg1_blockdim.y ? 1 : 0);

    std::cout << "About to run with stg1_griddim = (" << stg1_griddim.x << " x " << stg1_griddim.y
              << ") and stg1_blockdim = (" << stg1_blockdim.x << " x " << stg1_blockdim.y << ") thread blocks\n";

    float3* d_output = nullptr;
    // Malloc n_arrays * n_sums (which is stg1_griddim.x) elements
    hipMalloc (&d_output, n_arrays * stg1_griddim.x * 3 * sizeof(float));

    reduceit_arrays<<<stg1_griddim, stg1_blockdim>>>(in, d_output, n_arrays, n_elements);
    hipDeviceSynchronize();

    float3* d_final = nullptr;
    // Malloc n_arrays elements for the final sums (or could re-use d_output)
    hipMalloc (&d_final, n_arrays * 3 * sizeof(float));

    // stg1_griddim.x is n_sums
    sums.resize (stg1_griddim.x * n_arrays, make_float3(0,0,0));
    std::cout << "resized sums to have size " << stg1_griddim.x << " * " << n_arrays << " = " << (stg1_griddim.x * n_arrays) << std::endl;
    // Copy intermediate d_output into sums
    hipMemcpy (sums.data(), d_output, sums.size() * 3 * sizeof(float), hipMemcpyDeviceToHost);

    // stg1_griddim.x is 'n_sums'
    warps_base = stg1_griddim.x / 32;
    warps_extra = stg1_griddim.x % 32;
    tbx = (warps_base * 32) + (warps_extra ? 32 : 0);
    tbx = std::min (tbx, threadsperblock);
    dim3 stg2_blockdim(tbx, 1);
    dim3 stg2_griddim(1, 1);
    stg2_griddim.x = stg1_griddim.x / stg1_blockdim.x + (stg1_griddim.x % stg2_blockdim.x ? 1 : 0);
    stg2_griddim.y = n_arrays / stg2_blockdim.y + (n_arrays % stg2_blockdim.y ? 1 : 0);

    reduceit_arrays<<<stg2_griddim, stg2_blockdim>>>(d_output, d_final, n_arrays, stg1_griddim.x);
    // out_final can be only n_arrays in size

    final_sums.resize (n_arrays, make_float3(0,0,0));
    std::cout << "resized final sums to have size " << n_arrays << " = " << n_arrays << std::endl;
    // Copy intermediate d_output into sums
    hipMemcpy (final_sums.data(), d_final, final_sums.size() * 3 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree (d_output);
    hipFree (d_final);
}

int main()
{
    int arraysz = elements_per_array * number_of_arrays;

    std::vector<std::vector<float3>> many_arrays (number_of_arrays);

    // This is the equivalent of the values in GPU ram that I need to average
    std::vector<float3> many_arrays_in_seq (arraysz);

    float v1 = 0.0f;
    float v2 = 0.0f;
    float v3 = 0.0f;
    for (int j = 0; j < number_of_arrays; ++j) {
        many_arrays[j].resize (elements_per_array, make_float3(0,0,0));
        float e = j % 2 == 0 ? j* 10.0f : -j* 10.0f;
        //std::cout << "For array " << j << " extra data = " << e << std::endl;
        for (int i = 0; i < elements_per_array; ++i) {
            if (i % 2 == 0) {
               many_arrays[j][i] = make_float3 (v1+e, -v2+e, v3+e);
            } else {
                many_arrays[j][i] = make_float3 (-v1+e, v2+e, -v3+e);
            }
            many_arrays_in_seq[j * elements_per_array + i] = many_arrays[j][i];
        }
    }

#if 1
    for (int j = 0; j < number_of_arrays; ++j) {
        float3 cpu_sum = make_float3 (0.0f, 0.0f, 0.0f);
        for (int i = 0; i < elements_per_array; ++i) {
            cpu_sum.x += many_arrays[j][i].x;
            cpu_sum.y += many_arrays[j][i].y;
            cpu_sum.z += many_arrays[j][i].z;
        }
        std::cout << "cpu_sum for array " << j << " is (" << cpu_sum.x << "," << cpu_sum.y << "," << cpu_sum.z << ")\n";
    }
#endif

    // Copy to GPU memory:
    float3* d_many_arrays = nullptr;
    hipMalloc (&d_many_arrays, arraysz * 3 * sizeof(float));
    hipMemcpy (d_many_arrays, many_arrays_in_seq.data(), arraysz * 3 * sizeof(float), hipMemcpyHostToDevice);

    std::vector<float3> intermediate_sums; // container for intermediates (debug)
    std::vector<float3> gpu_sums; // container for the final sums
    shufflesum_arrays (d_many_arrays, number_of_arrays, elements_per_array, intermediate_sums, gpu_sums); // resize gpu_sums in here

    int num_sums = gpu_sums.size() / number_of_arrays;
    for (int i = 0; i < number_of_arrays; ++i) {
        std::cout << "\nGPU array i=" << i << " sums are: ";
        for (int j = 0; j < num_sums; ++j) {
            int idx = i * num_sums + j;
            std::cout << "(" << gpu_sums[idx].x << "," << gpu_sums[idx].y << "," << gpu_sums[idx].z << "), ";
        }
        std::cout << std::endl;
    }

    return 0;
}
